#include "hip/hip_runtime.h"
/**
 * Assignment 03 Program 
 * Sarah Helble
 * 9/17/17
 *
 * TODO: add description of what it does
 */

#include <stdio.h>
 
#define ARRAY_SIZE 256
#define ARRAY_SIZE_IN_BYTES (sizeof(unsigned int) * (ARRAY_SIZE))
#define MIN_ALPHA 32;
#define MAX_ALPHA 126;
 
 
char cpu_text[ARRAY_SIZE]; 
char cpu_key[ARRAY_SIZE];
 
__global__
 
void encrypt(char *text, char *key)
{
 		/* Calculate the current index */
 		const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
 
 		/* Create the cipherchar (addition of plaintext char and key char */
 		char cipherchar = ( text[thread_idx] + key[thread_idx] );
 
 		/* Make sure you're within the set of printable characters */
		if(cipherchar > MAX_ALPHA) { 
 			cipherchar = (cipherchar - MAX_ALPHA) + ( MIN_ALPHA - 1 );
 		}
 	
 		/* Save back in text array */
 		text[thread_idx] = cipherchar; 
}
 
void main_sub()
{
 		printf("Encrypting text: \n");
 		for(int i = 0; i < ARRAY_SIZE; i++) {
        		printf("%c", cpu_text[i]);
        }
                                      
 		printf("With Key: \n");	
 		for(int i = 0; i < ARRAY_SIZE; i++) {
 				printf("%c", cpu_key[i]);
        }
 
 		/* Declare and allocate pointers for GPU based parameters */
 		char *gpu_text;
 		char *gpu_key;
 
 		hipMalloc((void **)&gpu_text, ARRAY_SIZE_IN_BYTES);
 		hipMalloc((void **)&gpu_key, ARRAY_SIZE_IN_BYTES);
 		
 		/* Copy the CPU memory to the GPU memory */
 		hipMemcpy( cpu_text, gpu_text, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
 		hipMemcpy( cpu_key, gpu_key, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
 
 		/* Designate the number of blocks and threads */
 		const unsigned int num_blocks = ARRAY_SIZE/16;
 		const unsigned int num_threads = ARRAY_SIZE/num_blocks;
 
 		/* Execute the encryption kernel */
 		encrypt<<<num_blocks, num_threads>>>(gpu_text, gpu_key);
 
 		/* Copy the GPU memory back to the CPU */
 		hipMemcpy( cpu_text, gpu_text, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
 		hipMemcpy( cpu_key, gpu_key, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
 
 		/* Free the GPU memory */
 		hipFree(gpu_text);
 		hipFree(gpu_key);
 
 		/* Print the final result */
 		                                               
        printf("Results in ciphertext: \n");
        for (int i = 0; i < ARRAY_SIZE; i++) {
 				printf("%c", cpu_text[i]);
 		}
                                      
 }
 
 int main()
 {
 		/* TODO: get input file, array size and num blocks from command line */
 
 		/* TODO: Change this to read from file */
        FILE *input_fp = fopen("input_text.txt", "r");
        for(int i = 0; i < ARRAY_SIZE; i++) {
 				cpu_text[i] = (char) toupper(fgetc(input_fp));
 				cpu_key[i] = 'A';  // TODO: Make key random
 		}
 
 		main_sub();
 	
 		return EXIT_SUCCESS;
 }
 