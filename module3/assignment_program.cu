#include "hip/hip_runtime.h"
/**
 * Assignment 03 Program 
 * Sarah Helble
 * 9/17/17
 *
 * Command line args <total_num_threads> <threads_per_block> <input_file> <key_file>
 *
 * Creates two arrays of <total_num_threads> length, and reads <total_num_threads> 
 * characters from <input_file> and <key_file> to fill them.
 * Adds the character values together to create a cipher text (caesar cipher with
 * keyword)
 * 
 * Uses <total_num_threads> as total number of threads for the execution. 
 * Creates blocks with <threads_per_block> each. 
 * This results in # blocks = <total_num_threads> / <threads_per_block>
 */

#include <stdio.h>
#include <stdlib.h>

/* Number of characters in the alphabet */
#define NUM_ALPHA 127

/**
 * Kernel function that creates a ciphertext by adding the values 
 * in @text to the values in @key. As in a caesar cipher with keyword.
 *
 * @text plaintext values
 * @key key values
 * @result ciphertext
 *
 * TODO: some of the values in the resultant ciphertext are unprintable.
 * Make wrap around more advanced to deal with this. 
 */
__global__ void encrypt(unsigned int *text, unsigned int *key, unsigned int *result)
{
 	/* Calculate the current index */
 	const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
 
 	/* Create the cipherchar (addition of plaintext char and key char */
 	result[idx] = (unsigned int) ( ( key[idx] + text[idx] ) % NUM_ALPHA );

 	/* TODO: Some of these values are unprintable. Make wrap more advanced */
}

/**
 * Function that sets up everything for the kernel function encrypt()
 * 
 * @array_size size of array (total number of threads)
 * @threads_per_block number of threads to put in each block
 * @input_fp file pointer to the input file text 
 * @key_fp file pointer to the key file
 * 
 * Closes the file pointers @input_fp and @key_fp
 */
void main_sub(int array_size, int threads_per_block, FILE *input_fp, FILE *key_fp)
{
	/* Calculate the size of the array */
 	int array_size_in_bytes = (sizeof(unsigned int) * (array_size));
 	int i = 0;

 	unsigned int cpu_text[array_size]; 
	unsigned int cpu_key[array_size];
	unsigned int cpu_result[array_size];

	/* Read characters from the input and key files into the text and key arrays respectively */
        for(i = 0; i < array_size; i++) {
 		cpu_text[i] = fgetc(input_fp);
 		cpu_key[i] = fgetc(key_fp);  
	}
	
	/* Close the file pointers */
        fclose(input_fp);
        fclose(key_fp);
 
	/* Print the plain text and the key */
 	printf("Encrypting text: \n");
 	for(i = 0; i < array_size; i++) {
        	printf("%c", cpu_text[i]);
        }
 	printf("\n With Key: \n");    
 	for(i = 0; i < array_size; i++) {
 		printf("%c", cpu_key[i]);
        }
 
 	/* Declare and allocate pointers for GPU based parameters */
 	unsigned int *gpu_text;
 	unsigned int *gpu_key;
 	unsigned int *gpu_result
 	hipMalloc((void **)&gpu_text, array_size_in_bytes);
 	hipMalloc((void **)&gpu_key, array_size_in_bytes);
	hipMalloc((void **)&gpu_result, array_size_in_bytes);
 		
 	/* Copy the CPU memory to the GPU memory */
 	hipMemcpy( gpu_text, cpu_text, array_size_in_bytes, hipMemcpyHostToDevice);
 	hipMemcpy( gpu_key, cpu_key, array_size_in_bytes, hipMemcpyHostToDevice);
	
 	/* Designate the number of blocks and threads */
 	const unsigned int num_blocks = array_size/threads_per_block;
 	const unsigned int num_threads = array_size/num_blocks;
 
 	/* Execute the encryption kernel */
 	encrypt<<<num_blocks, num_threads>>>(gpu_text, gpu_key, gpu_result);
 
 	/* Copy the GPU memory back to the CPU */
	hipMemcpy( cpu_result, gpu_result, array_size_in_bytes, hipMemcpyDeviceToHost);
 
 	/* Free the GPU memory */
 	hipFree(gpu_text);
 	hipFree(gpu_key);
	hipFree(gpu_result);
 
 	/* Print the resulting ciphertext */
        printf("\nResults in ciphertext: \n");
        for(i = 0; i < array_size; i++) {
 		printf("%c ", (int)cpu_result[i]);	
 	}
 	printf("\n");                                     
 }
 
int main(int argc, char *argv[])
{
	/* Check the number of arguments, print usage if wrong */
	if(argc != 5) {
        	printf("Error, usage: %s <total_num_threads> <threads_per_block> <input_file> <key_file>\n", argv[0]);
		exit(-1);
        }
        
        int num_threads = atoi(argv[1]);
        int threads_per_block = atoi(argv[2]);
        char *input_filename = argv[3];       
        char *key_filename = argv[4];
        
	/* Make sure the input text file and the key file are openable */
        FILE *input_fp = fopen(input_filename, "r");
        if(!input_fp) {
		printf("Error: failed to open input file %s\n", argv[3]);
		exit(-1);
        }
        FILE *key_fp = fopen(key_filename, "r");
        if(!key_fp){
		printf("Error: failed to open key file %s\n", argv[4]);
		fclose(input_fp);
		exit(-1);
        }
		
	/* Pass all arguments to the subordinate main function */
 	main_sub(num_threads, threads_per_block, input_fp, key_fp);
	
        return EXIT_SUCCESS;
 }
 
