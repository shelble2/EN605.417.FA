#include "hip/hip_runtime.h"
/**
 * Assignment 08
 * Beginnings of what would be needed to produce random sudoku puzzles.
 * This program produces a square matrix and fills each cell with a random
 * value between 1 and MAX_INT (inclusive).
 *
 * Future work will make this production follow the rules of sudoku (i.e., one
 * of each value in each row, col, square)
 *
 * Sarah Helble
 * 22 Oct 2017
 *
 **/

#include <unistd.h>
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define MAX_INT 9               // 9 is standard sudoku
#define CELLS MAX_INT * MAX_INT // sudokus are square 9 x 9

#define THREADS_PER_BLOCK 1

/**
 * This kernel initializes the states for each input in the array
 * @seed is the seed for the init function
 * @states is an allocated array, where the output of this fuction will be stored
 */
__global__ void init_states(unsigned int seed, hiprandState_t* states) {
  /* Calculate the current index */
  const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  hiprand_init(seed, idx, 0, &states[idx]);
}

/**
 * Given the passed array of states @states, this kernel fills allocated array
 * @numbers with a random int between 0 and MAX_INT
 * @states is the set of states already initialized by CUDA
 * @numbers is an allocated array where this kernel function will put its output
 */
__global__ void fill_grid(hiprandState_t* states, unsigned int* numbers) {
  /* Calculate the current index */
  const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  numbers[idx] = hiprand(&states[idx]) % MAX_INT;

  // If we got a 0, make it MAX_INT, since sudokus don't have 0's
  //TODO: think of a more elegant way to do this.
  if(numbers[idx] == 0) {
      numbers[idx] = MAX_INT;
  }
}

/**
 * Prints the passed array like a sudoku puzzle in ascii art
 * @numbers array to print
 */
 void sudoku_print(unsigned int* numbers)
{
  int i;
  int j;
  int block_dim = round(sqrt(MAX_INT));

  printf("\n_________________________________________\n");

  for (i = 0; i < MAX_INT; i++) {

    printf("||");
    for (j = 0; j < MAX_INT; j++) {
      printf(" %u |", numbers[ ( (i*MAX_INT) + j ) ]);
      if((j+1) % block_dim == 0) {
          printf("|");
      }
    }

    j = 0;
    //Breaks between each row
    if( ((i+1) % block_dim) == 0) {
      printf("\n||___|___|___||___|___|___||___|___|___||\n");
    } else {
      //TODO:make this able to handle other sizes prettily
      printf("\n||---|---|---||---|---|---||---|---|---||\n");
    }
 }
}

void main_sub( ) {
  //TODO: Put it in 2D array or matrix
  //TODO: add timing data

  const unsigned int num_blocks = CELLS/THREADS_PER_BLOCK;
  const unsigned int num_threads = CELLS/num_blocks;

  hipEvent_t start, stop;
	float duration;

	hipEventCreate(&start);
	hipEventCreate(&stop);

  /* Recording from init to copy back */
	hipEventRecord(start, 0);

  hiprandState_t* states;
  hipMalloc((void**) &states, CELLS * sizeof(hiprandState_t));

  /* invoke the GPU to initialize the states for cuRAND */
  init_states<<<num_blocks, num_threads>>>(time(0), states);

  unsigned int* nums;
  hipHostMalloc((void**) &nums, CELLS * sizeof(unsigned int));

  unsigned int* d_nums;
  hipMalloc((void**) &d_nums, CELLS * sizeof(unsigned int));

  /* invoke the kernel to get some random numbers */
  fill_grid<<<num_blocks, num_threads>>>(states, d_nums);

  /* copy the result back to the CPU */
  hipMemcpy(nums, d_nums, CELLS * sizeof(unsigned int), hipMemcpyDeviceToHost);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&duration, start, stop);
  printf("Elapsed Time: %f", duration);

  sudoku_print(nums);

  /* free the memory we allocated for the states and numbers */
  hipFree(states);
  hipFree(d_nums);
  hipFree(nums);
}

/**
 * Starting here so that we can easily execute two runs of each kernel without
 * modifying surrounding functions
 */
int main() {
  int iters = 2;

  for(int i = 0; i < iters; i++) {
    printf("\nRun #%d of kernel function:\n", i+1);
    main_sub();
  }

  return 0;
}
