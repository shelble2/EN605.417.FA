#include "hip/hip_runtime.h"
/**
 * Assignment 08
 * Beginnings of what would be needed to produce random sudoku puzzles.
 * This program produces a square matrix and fills each cell with a random
 * value between 1 and MAX_INT (inclusive).
 *
 * Future work will make this production follow the rules of sudoku (i.e., one
 * of each value in each row, col, square)
 *
 * Sarah Helble
 * 22 Oct 2017
 *
 **/

#include <unistd.h>
#include <stdio.h>
#include <math.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define MAX_INT 9               // 9 is standard sudoku
#define CELLS MAX_INT * MAX_INT // sudokus are square 9 x 9

#define THREADS_PER_BLOCK 1

/**
 * This kernel initializes the states for each input in the array
 * @seed is the seed for the init function
 * @states is an allocated array, where the output of this fuction will be stored
 */
__global__ void init_states(unsigned int seed, hiprandState_t* states) {
  /* Calculate the current index */
  const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  hiprand_init(seed, idx, 0, &states[idx]);
}

/**
 * Given the passed array of states @states, this kernel fills allocated array
 * @numbers with a random int between 0 and MAX_INT
 * @states is the set of states already initialized by CUDA
 * @numbers is an allocated array where this kernel function will put its output
 */
__global__ void fill_grid(hiprandState_t* states, unsigned int* numbers) {
  /* Calculate the current index */
  const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  numbers[idx] = hiprand(&states[idx]) % MAX_INT;

  // If we got a 0, make it MAX_INT, since sudokus don't have 0's
  //TODO: think of a more elegant way to do this.
  if(numbers[idx] == 0) {
      numbers[idx] = MAX_INT;
  }
}

/**
 * Prints the passed array like a sudoku puzzle in ascii art
 * @numbers array to print
 */
 //TODO: need to add block breaks
 void sudoku_print(unsigned int* numbers)
{
  int i;
  int j;
  float block_dim = round(sqrt(MAX_INT));

  printf("\n________________________________________________________");

  for (i = 0; i < MAX_INT; i++) {
    //Breaks between each row
    if(i % block_dim == 0) {
      printf("\n____________________________________________________________\n");
    } else if(i != 0) {
      printf("\n------------------------------------------------------\n");
    }

    // Between each cell
    printf("|");
    for (j = 0; j < MAX_INT; j++) {
      printf(" %u |", numbers[ ( (i*MAX_INT) + j ) ]);
      if(j % block_dim == 0) {
          printf("|");
      }
    }

    j = 0;
  }

  printf("\n________________________________________________________\n");
}

int main( ) {
  //TODO: make it print like a sudoku. Put it in 2D array

   /* CUDA's random number library uses hiprandState_t to keep track
      of the seed value
      we will store a random state for every thread  */
  hiprandState_t* states;

  /* allocate space on the GPU for the random states */
  hipMalloc((void**) &states, CELLS * sizeof(hiprandState_t));

  const unsigned int num_blocks = CELLS/THREADS_PER_BLOCK;
  const unsigned int num_threads = CELLS/num_blocks;

  /* invoke the GPU to initialize all of the random states */
  init_states<<<num_blocks, num_threads>>>(time(0), states);

  /* allocate an array of unsigned ints on the CPU and GPU */
  unsigned int cpu_nums[CELLS];
  unsigned int* gpu_nums;
  hipMalloc((void**) &gpu_nums, CELLS * sizeof(unsigned int));

  /* invoke the kernel to get some random numbers */
  fill_grid<<<num_blocks, num_threads>>>(states, gpu_nums);

  /* copy the random numbers back */
  hipMemcpy(cpu_nums, gpu_nums, CELLS * sizeof(unsigned int), hipMemcpyDeviceToHost);

  sudoku_print(cpu_nums);

  /* free the memory we allocated for the states and numbers */
  hipFree(states);
  hipFree(gpu_nums);

  return 0;
}
