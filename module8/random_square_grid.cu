#include "hip/hip_runtime.h"
/**
 * Assignment 08
 * Beginnings of what would be needed to produce random sudoku puzzles.
 * This program produces a square matrix and fills each cell with a random
 * value between 1 and MAX_INT (inclusive).
 *
 * In order to try out a second CUDA library, this program also uses cuBLAS to
 * invert the resulting matrix
 *
 * Future work will make this production follow the rules of sudoku (i.e., one
 * of each value in each row, col, square)
 *
 * Sarah Helble
 * 22 Oct 2017
 *
 **/

#include <unistd.h>
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define MAX_INT 9               // 9 is standard sudoku
#define CELLS MAX_INT * MAX_INT // sudokus are square 9 x 9

#define THREADS_PER_BLOCK 1

/**
 * This kernel initializes the states for each input in the array
 * @seed is the seed for the init function
 * @states is an allocated array, where the output of this fuction will be stored
 */
__global__ void init_states(unsigned int seed, hiprandState_t* states) {
  /* Calculate the current index */
  const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  hiprand_init(seed, idx, 0, &states[idx]);
}

/**
 * Given the passed array of states @states, this kernel fills allocated array
 * @numbers with a random int between 0 and MAX_INT
 * @states is the set of states already initialized by CUDA
 * @numbers is an allocated array where this kernel function will put its output
 */
__global__ void fill_grid(hiprandState_t* states, unsigned int* numbers) {
  /* Calculate the current index */
  const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  numbers[idx] = hiprand(&states[idx]) % MAX_INT;

  // If we got a 0, make it MAX_INT, since sudokus don't have 0's
  //TODO: think of a more elegant way to do this.
  if(numbers[idx] == 0) {
      numbers[idx] = MAX_INT;
  }
}

/**
 * Multiply the two passed matrices into the resultant matrix
 * @A and @B are the matrices to Multiply
 * @result is the result
 */
__global__ void matrix_multiply(unsigned int *A, unsigned int *B, unsigned int *result) {
  /* Calculate the current index */
  const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  //TODO: actually multiply here instead of just copying A
  result[idx] = A[idx];
}

/**
 * Prints the passed array like a sudoku puzzle in ascii art
 * @numbers array to print
 */
 void sudoku_print(unsigned int* numbers)
{
  int i;
  int j;
  int block_dim = round(sqrt(MAX_INT));

  printf("\n_________________________________________\n");

  for (i = 0; i < MAX_INT; i++) {

    printf("||");
    for (j = 0; j < MAX_INT; j++) {
      printf(" %u |", numbers[ ( (i*MAX_INT) + j ) ]);
      if((j+1) % block_dim == 0) {
          printf("|");
      }
    }

    j = 0;
    //Breaks between each row
    if( ((i+1) % block_dim) == 0) {
      printf("\n||___|___|___||___|___|___||___|___|___||\n");
    } else {
      //TODO:make this able to handle other sizes prettily
      printf("\n||---|---|---||---|---|---||---|---|---||\n");
    }
 }
}

/**
 * Harness for the creation of random nxn matrices
 * Returns matrix of unsigned ints to be freed by caller
 */
void rand_sub(unsigned int **out) {
  const unsigned int num_blocks = CELLS/THREADS_PER_BLOCK;
  const unsigned int num_threads = CELLS/num_blocks;

  hiprandState_t* states;
  unsigned int *tmp, *nums, *d_nums;

  hipEvent_t start, stop;
	float duration;

	hipEventCreate(&start);
	hipEventCreate(&stop);

  tmp = (unsigned int*)malloc(CELLS * (sizeof(unsigned int)));
  hipHostMalloc((void**) &nums, CELLS * sizeof(unsigned int));
  hipMalloc((void**) &states, CELLS * sizeof(hiprandState_t));
  hipMalloc((void**) &d_nums, CELLS * sizeof(unsigned int));

  /* Recording from init to copy back */
	hipEventRecord(start, 0);

  /* Allocate space and invoke the GPU to initialize the states for cuRAND */
  init_states<<<num_blocks, num_threads>>>(time(0), states);

  /* invoke the kernel to generate random numbers */
  fill_grid<<<num_blocks, num_threads>>>(states, d_nums);

  /* copy the result back to the CPU */
  hipMemcpy(nums, d_nums, CELLS * sizeof(unsigned int), hipMemcpyDeviceToHost);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&duration, start, stop);
  printf("Elapsed Time: %f", duration);

  sudoku_print(nums);

  /* free the memory we allocated for the states and numbers */
  hipFree(states);
  hipFree(d_nums);

  memcpy(tmp, nums, CELLS *sizeof(unsigned int));

  hipHostFree(nums);

  *out = tmp;
}

/**
 * Sub function for handling calls to the matrix multiplication
 * kernel function
 * @matrix_1 and @matrix_2 are two matrices to multiply together
 */
void blas_sub(unsigned int *matrix_1, unsigned int *matrix_2)
{
  const unsigned int num_blocks = CELLS/THREADS_PER_BLOCK;
  const unsigned int num_threads = CELLS/num_blocks;
  const unsigned int array_size_in_bytes = CELLS *sizeof(unsigned int);

  unsigned int *m_A, *m_B, *result, *d_A, *d_B, *d_result;

  hipEvent_t start, stop;
  float duration;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipHostMalloc((void**) &m_A, array_size_in_bytes);
  hipHostMalloc((void**) &m_B, array_size_in_bytes);
  hipHostMalloc((void**) &result, array_size_in_bytes);

  //Copy passed arrays to pinned memory
  memcpy(m_A, matrix_1, array_size_in_bytes);
  memcpy(m_B, matrix_2, array_size_in_bytes);

  hipMalloc((void**) &d_A, array_size_in_bytes);
  hipMalloc((void**) &d_B, array_size_in_bytes);
  hipMalloc((void**) &d_result, array_size_in_bytes);

  //copy pinned host memory to device memory
  hipMemcpy( d_A, m_A, array_size_in_bytes, hipMemcpyHostToDevice);
  hipMemcpy( d_B, m_B, array_size_in_bytes, hipMemcpyHostToDevice);

  /* Recording from init to copy back */
  hipEventRecord(start, 0);

  /* Allocate space and invoke the GPU to initialize the states for cuRAND */
  matrix_multiply<<<num_blocks, num_threads>>>(d_A, d_B, d_result);

  //Copy the result back to the host
  hipMemcpy(result, d_result, array_size_in_bytes, hipMemcpyDeviceToHost);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&duration, start, stop);
  printf("Elapsed Time: %f", duration);

  sudoku_print(result);

  hipHostFree(m_A);
  hipHostFree(m_B);
  hipHostFree(result);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_result);
}

/**
 * Starting here so that we can easily execute two runs of each kernel without
 * modifying surrounding functions
 */
int main() {
  unsigned int *A, *B, *C, *D;

  printf("\nRun #1 of cuRAND kernel function. Matrix A:\n");
  rand_sub(&A);
  printf("\n");

  printf("\nRun #2 of cuRAND kernel function. Matrix B:\n");
  rand_sub(&B);
  printf("\n");

  printf("\nRun #3 of cuRAND kernel function. Matrix C:\n");
  rand_sub(&C);
  printf("\n");

  printf("\nRun #4 of cuRAND kernel function. Matrix D:\n");
  rand_sub(&D);
  printf("\n");

  printf("\nRun #1 of cuBLAS kernel function. Matrix A x Matrix B:\n");
  blas_sub(A, B);

  printf("\nRun #2 of cuBLAS kernel function. Matrix C x Matrix D:\n");
  blas_sub(C, D);

  free(A);
  free(B);
  free(C);
  free(D);
  return 0;
}
