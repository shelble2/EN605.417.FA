/**
 * Assignment 05 Program 
 * Sarah Helble
 * 9/29/17
 *
 * Usage ./out <total_num_threads> <threads_per_block>
 *
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/**
 * Returns the current time
 */
__host__ hipEvent_t get_time(void)
{
	hipEvent_t time;
	hipEventCreate(&time);
	hipEventRecord(time);
	return time;
}

/**
 * Kernel function that shuffles the values in @ordered and puts the
 * output in @shuffled
 */
__global__ void shuffle(unsigned int *ordered, unsigned int *shuffled)
{
  /* Calculate the current index */
  const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  shuffled[idx] = 100;
}

/**
 * One fuction to handle the printing of results.
 * @ordered is the original array
 * @shuffled is the result 
 */
void print_results(unsigned int *ordered, unsigned int *shuffled, int array_size)
{
  int i = 0;

  printf("\n");
  for(i = 0; i < array_size; i++) {
    printf("Original value at index [%d]: %d, shuffled: %d\n", i, ordered[i], shuffled[i]);
  }
  printf("\n");
}

/**
 * Function that sets up everything for the kernel function 
 * with simple pageable host memory
 *
 * @array_size size of array (total number of threads)
 * @threads_per_block number of threads to put in each block
 */
void pageable_transfer_execution(int array_size, int threads_per_block)
{
  /* Calculate the size of the array */
  int array_size_in_bytes = (sizeof(unsigned int) * (array_size));
  int i = 0;

  unsigned int *ordered = (unsigned int *) malloc(array_size_in_bytes);
  unsigned int *shuffled_result = (unsigned int *) malloc(array_size_in_bytes);

  // Fill the ordered array
  for(i = 0; i < array_size; i++) {
  	ordered[i] = i;	
  }

  /* Declare and allocate pointers for GPU based parameters */
  unsigned int *d_ordered;
  unsigned int *d_shuffled_result;

  hipMalloc((void **)&d_ordered, array_size_in_bytes);
  hipMalloc((void **)&d_shuffled_result, array_size_in_bytes);

  /* Copy the CPU memory to the GPU memory */
  hipMemcpy( d_ordered, ordered, array_size_in_bytes, hipMemcpyHostToDevice);

  /* Designate the number of blocks and threads */
  const unsigned int num_blocks = array_size/threads_per_block;
  const unsigned int num_threads = array_size/num_blocks;

  /* Execute the encryption kernel and keep track of start and end time for duration */
  float duration = 0;
  hipEvent_t start_time = get_time();

  shuffle<<<num_blocks, num_threads>>>(d_ordered, d_shuffled_result);

  hipEvent_t end_time = get_time();
  hipEventSynchronize(end_time);
	hipEventElapsedTime(&duration, start_time, end_time);

  /* Copy the changed GPU memory back to the CPU */
  hipMemcpy( shuffled_result, d_shuffled_result, array_size_in_bytes, hipMemcpyDeviceToHost);

  printf("Pageable Transfer- Duration: %fmsn\n", duration);
  print_results(ordered, shuffled_result, array_size);

  /* Free the GPU memory */
  hipFree(d_ordered);
  hipFree(d_shuffled_result);

  /* Free the CPU memory */
  free(ordered);
  free(shuffled_result);
}

/**
 * Function that sets up everything for the kernel function encrypt()
 *
 * @array_size size of array (total number of threads)
 * @threads_per_block number of threads to put in each block
 * @input_fp file pointer to the input file text
 * @key_fp file pointer to the key file
 *
 * Closes the file pointers @input_fp and @key_fp
 */
void pinned_transfer_execution(int array_size, int threads_per_block, FILE *input_fp, FILE *key_fp)
{
  /* Calculate the size of the array */
  int array_size_in_bytes = (sizeof(unsigned int) * (array_size));
  int i = 0;

  /*host pageable */
  unsigned int *cpu_text_pageable = (unsigned int *) malloc(array_size_in_bytes);
  unsigned int *cpu_key_pageable = (unsigned int *) malloc(array_size_in_bytes);
  unsigned int *cpu_result_pageable = (unsigned int *) malloc(array_size_in_bytes);

  /* Read characters from the input and key files into the text and key arrays respectively */
  for(i = 0; i < array_size; i++) {
    cpu_text_pageable[i] = fgetc(input_fp);
    cpu_key_pageable[i] = fgetc(key_fp);
    if(feof(input_fp) || feof(key_fp)) {
        rewind(input_fp);
        rewind(key_fp);
    }
  }

  //host pinned
  unsigned int *cpu_text_pinned;
  unsigned int *cpu_key_pinned;
  unsigned int *cpu_result_pinned;

  //pin it
  hipHostMalloc((void **)&cpu_text_pinned, array_size_in_bytes, hipHostMallocDefault);
  hipHostMalloc((void **)&cpu_key_pinned, array_size_in_bytes, hipHostMallocDefault);
  hipHostMalloc((void **)&cpu_result_pinned, array_size_in_bytes, hipHostMallocDefault);

  /* Copy the memory over */
  memcpy(cpu_text_pinned, cpu_text_pageable, array_size_in_bytes);
  memcpy(cpu_key_pinned, cpu_key_pageable, array_size_in_bytes);
  memcpy(cpu_result_pinned, cpu_result_pageable, array_size_in_bytes);

  /* Declare and allocate pointers for GPU based parameters */
  unsigned int *gpu_text;
  unsigned int *gpu_key;
  unsigned int *gpu_result;

  hipMalloc((void **)&gpu_text, array_size_in_bytes);
  hipMalloc((void **)&gpu_key, array_size_in_bytes);
  hipMalloc((void **)&gpu_result, array_size_in_bytes);

  /* Copy the CPU memory to the GPU memory */
  hipMemcpy( gpu_text, cpu_text_pinned, array_size_in_bytes, hipMemcpyHostToDevice);
  hipMemcpy( gpu_key, cpu_key_pinned, array_size_in_bytes, hipMemcpyHostToDevice);

  /* Designate the number of blocks and threads */
  const unsigned int num_blocks = array_size/threads_per_block;
  const unsigned int num_threads = array_size/num_blocks;

  /* Execute the encryption kernel and keep track of start and end time for duration */
  float duration = 0;
  hipEvent_t start_time = get_time();

  shuffle<<<num_blocks, num_threads>>>(gpu_text, gpu_result);

  hipEvent_t end_time = get_time();
  hipEventSynchronize(end_time);
	hipEventElapsedTime(&duration, start_time, end_time);

  /* Copy the changed GPU memory back to the CPU */
  hipMemcpy( cpu_result_pinned, gpu_result, array_size_in_bytes, hipMemcpyDeviceToHost);

  printf("Pinned Transfer- Duration: %fmsn\n", duration);
  print_results(cpu_text_pinned, cpu_result_pinned, array_size);

  /* Free the GPU memory */
  hipFree(gpu_text);
  hipFree(gpu_key);
  hipFree(gpu_result);

  /* Free the pinned CPU memory */
  hipHostFree(cpu_text_pinned);
  hipHostFree(cpu_key_pinned);
  hipHostFree(cpu_result_pinned);

  /* Free the pageable CPU memory */
  free(cpu_text_pageable);
  free(cpu_key_pageable);
  free(cpu_result_pageable);

}

/**
 * Prints the correct usage of this file
 * @name is the name of the executable (argv[0])
 */
void print_usage(char *name)
{
  printf("Usage: %s <total_num_threads> <threads_per_block> <input_file> <key_file>\n", name);
}

/**
 * Performs simple setup functions before calling the pageable_transfer_execution()
 * function.
 * Makes sure the files are valid, handles opening and closing of file pointers.
 */
void pageable_transfer(int num_threads, int threads_per_block, char *input_file, char *key_file)
{
  /* Make sure the input text file and the key file are openable */
  FILE *input_fp = fopen(input_file, "r");
  if(!input_fp) {
    printf("Error: failed to open input file %s\n", input_file);
    exit(-1);
  }
  FILE *key_fp = fopen(key_file, "r");
  if(!key_fp){
    printf("Error: failed to open key file %s\n", key_file);
    fclose(input_fp);
    exit(-1);
  }

  /* Perform the pageable transfer */
  pageable_transfer_execution(num_threads, threads_per_block);

  fclose(input_fp);
  fclose(key_fp);
}

/**
 * Performs setup functions before calling the pageable_transfer_execution()
 * function.
 * Makes sure the files are valid, handles opening and closing of file pointers.
 */
void pinned_transfer(int num_threads, int threads_per_block, char *input_file, char *key_file)
{
  /* Make sure the input text file and the key file are openable */
  FILE *input_fp = fopen(input_file, "r");
  if(!input_fp) {
    printf("Error: failed to open input file %s\n", input_file);
    exit(-1);
  }
  FILE *key_fp = fopen(key_file, "r");
  if(!key_fp){
    printf("Error: failed to open key file %s\n", key_file);
    fclose(input_fp);
    exit(-1);
  }

  /* Perform the pageable transfer */
  pinned_transfer_execution(num_threads, threads_per_block, input_fp, key_fp);

  fclose(input_fp);
  fclose(key_fp);
}

/**
 * Entry point for excution. Checks command line arguments and
 * opens input files, then passes execution to subordinate main_sub()
 */
int main(int argc, char *argv[])
{
  /* Check the number of arguments, print usage if wrong */
  if(argc != 5) {
    printf("Error: Incorrect number of command line arguments\n");
    print_usage(argv[0]);
    exit(-1);
  }

  /* Check the values for num_threads and threads_per_block */
  int num_threads = atoi(argv[1]);
  int threads_per_block = atoi(argv[2]);
  if(num_threads <= 0 || threads_per_block <= 0) {
    printf("Error: num_threads and threads_per_block must be integer > 0");
    print_usage(argv[0]);
    exit(-1);
  }

  if(threads_per_block > num_threads) {
      printf("Error: threads per block is greater than number of threads\n");
      print_usage(argv[0]);
      exit(-1);
  }

  printf("\n");
  /* Perform the pageable transfer */
  pageable_transfer(num_threads, threads_per_block, argv[3], argv[4]);

  printf("-----------------------------------------------------------------\n");

  /* Perform the pinned transfer */
  pinned_transfer(num_threads, threads_per_block, argv[3], argv[4]);

  return EXIT_SUCCESS;
}
