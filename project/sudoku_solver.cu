#include "hip/hip_runtime.h"
/**
 * sudoku_solver.cu
 * Sarah Helble
 * 2017-11-14
 *
 * In the process of adapting shuffle.cu from Module 5 assigment to work on
 * solving sudoku puzzles in the form of a string of numbers, where 0 indicates
 * an empty cell.
 *
 * Should compile with `$ nvcc sudoku_solver.cu -o sudoku_solver` and run with
 * `$ ./sudoku_solver`
 */
/*
Lessons so far
- sharing accross blocks
- debug prints
*/

#include <stdio.h>
#include <stdlib.h>

#define DIM 9             // Customary sudoku
#define B_DIM 3           // dimension of one sudoku block
#define CELLS DIM * DIM   // 81
#define THREADS_PER_BLOCK DIM // Seems like a nice way to split..

/**
 * Returns the current time
 */
__host__ hipEvent_t get_time(void)
{
	hipEvent_t time;
	hipEventCreate(&time);
	hipEventRecord(time);
	return time;
}

/**
 * Kernel function that solves based on last available. If only one number
 * can fit in a given cell, based on the contents of its row, column, and block;
 * then fill the cell with that value.
 */
__global__ void solve_by_possibility(unsigned int *ordered, unsigned int *solved)
{
	__shared__ unsigned int tmp[CELLS];

	const unsigned int my_cell_id = threadIdx.x;
	const unsigned int col = my_cell_id % DIM;
	const unsigned int row = (my_cell_id - col) / DIM;

	unsigned int possibilities[DIM+1] = {0,1,1,1,1,1,1,1,1,1};

	tmp[my_cell_id] = ordered[my_cell_id];

	// Only try to solve if cell is empty
	if(tmp[my_cell_id] != 0 ) {
		tmp[my_cell_id]  = tmp[my_cell_id];
	} else {
		// Go through all in the same row
		for(int i = row * DIM; i < ((row*DIM) + DIM); i++) {
			int current = tmp[i];
			possibilities[current] = 0;
		}

		//Go through all in the same column
		for(int i = 0; i < DIM ; i++) {
			int current = tmp[i*DIM+col];
			possibilities[current] = 0;
		}

		//Go through all in the same block
		int s_row = row - (row % B_DIM);
		int s_col = col - (col % B_DIM);
		for(int i = s_row; i < (s_row + B_DIM); i++) {
			for(int j = s_col; j < (s_col + B_DIM); j++) {
				int current = tmp[(i*DIM)+j];
				possibilities[current] = 0;
			}
		}

		int candidate = 0;

		// If only one possibility is left, use it
		for(int i = 0; i < DIM+1; i++) {
			if(possibilities[i] == 1) {
				if (candidate == 0) {
					candidate = i;
				} else {
					candidate = 0;
					break;
				}
			}
		}

		tmp[my_cell_id] = candidate;
	}

	__syncthreads();

	solved[my_cell_id] = tmp[my_cell_id];
}

/**
 * Prints the passed array like a sudoku puzzle in ascii art
 * @numbers array to print
 */
 void sudoku_print(unsigned int* numbers)
{
  int i;
  int j;
  int block_dim = round(sqrt(DIM));

  printf("\n_________________________________________\n");

  for (i = 0; i < DIM; i++) {

    printf("||");
    for (j = 0; j < DIM; j++) {
      printf(" %u |", numbers[ ( (i*DIM) + j ) ]);
      if((j+1) % block_dim == 0) {
          printf("|");
      }
    }

    j = 0;
    //Breaks between each row
    if( ((i+1) % block_dim) == 0) {
      printf("\n||___|___|___||___|___|___||___|___|___||\n");
    } else {
      //TODO:make this able to handle other sizes prettily
      printf("\n||---|---|---||---|---|---||---|---|---||\n");
    }
 }
}

int check_if_done()

void main_sub()
{
  /* Calculate the size of the array */
  int array_size_in_bytes = (sizeof(unsigned int) * (CELLS));

  unsigned int h_puzzle[CELLS] = {0,0,4,3,0,0,2,0,9,
								  0,0,5,0,0,9,0,0,1,
								  0,7,0,0,6,0,0,4,3,
								  0,0,6,0,0,2,0,8,7,
								  1,9,0,0,0,7,4,0,0,
								  0,5,0,0,8,3,0,0,0,
								  6,0,0,0,0,0,1,0,5,
								  0,0,3,5,0,8,6,9,0,
								  0,4,2,9,1,0,3,0,0};
  unsigned int *h_pinned_puzzle;
  unsigned int *h_solution;

  //pin it
  hipHostMalloc((void **)&h_pinned_puzzle, array_size_in_bytes);
  hipHostMalloc((void **)&h_solution, array_size_in_bytes);

  // Copy it to pinned memory
  memcpy(h_pinned_puzzle, h_puzzle, array_size_in_bytes);

  /* Declare and allocate pointers for GPU based parameters */
  unsigned int *d_puzzle;
  unsigned int *d_solution;

  hipMalloc((void **)&d_puzzle, array_size_in_bytes);
  hipMalloc((void **)&d_solution, array_size_in_bytes);

  /* Copy the CPU memory to the GPU memory */
  hipMemcpy(d_puzzle, h_pinned_puzzle, array_size_in_bytes, hipMemcpyHostToDevice);

  /* Designate the number of blocks and threads */
  const unsigned int num_blocks = CELLS/THREADS_PER_BLOCK;
  const unsigned int num_threads = CELLS/num_blocks;

  printf("Puzzle:\n");
  sudoku_print(h_puzzle);

  /* Execute the kernel and keep track of start and end time for duration */
  float duration = 0;

  hipEvent_t start_time = get_time();

  //SCH: used to be num_blocks, num_threads, but think all has to be on same block to share
  solve_by_possibility<<<1, CELLS>>>(d_puzzle, d_solution);

  hipEvent_t end_time = get_time();
  hipEventSynchronize(end_time);

  hipEventElapsedTime(&duration, start_time, end_time);

  /* Copy the changed GPU memory back to the CPU */
  hipMemcpy(h_solution, d_solution, array_size_in_bytes, hipMemcpyDeviceToHost);

  printf("Increment 1:\n");
  sudoku_print(h_solution);

  hipMemcpy(d_puzzle, h_solution, array_size_in_bytes, hipMemcpyHostToDevice);

  solve_by_possibility<<<1,CELLS>>>(d_puzzle, d_solution);

  hipMemcpy(h_solution, d_solution, array_size_in_bytes, hipMemcpyDeviceToHost);

  printf("Increment 2:\n");
  sudoku_print(h_solution);

  hipMemcpy(d_puzzle, h_solution, array_size_in_bytes, hipMemcpyHostToDevice);

  solve_by_possibility<<<1,CELLS>>>(d_puzzle, d_solution);

  hipMemcpy(h_solution, d_solution, array_size_in_bytes, hipMemcpyDeviceToHost);

  printf("Increment 3:\n");
  sudoku_print(h_solution);

  hipMemcpy(d_puzzle, h_solution, array_size_in_bytes, hipMemcpyHostToDevice);

  solve_by_possibility<<<1,CELLS>>>(d_puzzle, d_solution);

  hipMemcpy(h_solution, d_solution, array_size_in_bytes, hipMemcpyDeviceToHost);

  printf("Solution:\n");
  sudoku_print(h_solution);

  printf("\tSolved in: %fmsn\n", duration);

  /* Free the GPU memory */
  hipFree(d_puzzle);
  hipFree(d_solution);

  /* Free the pinned CPU memory */
  hipHostFree(h_pinned_puzzle);
  hipHostFree(h_solution);
}

/**
 * Prints the correct usage of this file
 * @name is the name of the executable (argv[0])
 */
void print_usage(char *name)
{
  printf("Usage: %s \n", name);
}

/**
 * Entry point for execution. Checks command line arguments
 * then passes execution to subordinate function
 */
int main(int argc, char *argv[])
{
  /* Check the number of arguments, print usage if wrong */
  if(argc != 1) {
    printf("Error: Incorrect number of command line arguments\n");
    print_usage(argv[0]);
    exit(-1);
  }

  printf("\n");

  main_sub();

  return EXIT_SUCCESS;
}
