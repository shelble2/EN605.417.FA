/**
 * sudoku_solver.cu
 * Sarah Helble
 * 2017-11-01
 *
 * In the process of adapting shuffle.cu from Module 5 assigment to work on
 * solving sudoku puzzles in the form of a string of numbers, where 0 indicates
 * an empty cell.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define DIM 9                 // Customary sudoku
#define CELLS DIM * DIM   // 81
#define THREADS_PER_BLOCK DIM // Seems like a nice way to split..

//TODO: Look up again rules about sharing data across blocks. They will have to share

/**
 * Returns the current time
 */
__host__ hipEvent_t get_time(void)
{
	hipEvent_t time;
	hipEventCreate(&time);
	hipEventRecord(time);
	return time;
}

/**
 * Kernel function that moves the values in @ordered to @shuffled
 */
__global__ void solve(unsigned int *ordered, unsigned int *shuffled)
{
	__shared__ unsigned int tmp[CELLS];
	const unsigned int row = threadIdx.x;
	const unsigned int col = blockIdx.x;

	// col = 0
	// row = 2
	//DIM = 3
	// col * DIM = 0 + row = 2
	// 1 2 3
	// 4 5 6
	// 7 8 9
	//
	const unsigned int my_cell_id = (col * DIM) + row;

	tmp[my_cell_id] = ordered[my_cell_id];

	// Only try to solve if cell is empty
//	if(tmp[my_cell_id] == 0) {
		//see if there is only one number that can fit in the cell, given row, column, and block entries
		tmp[my_cell_id] = 10;
	//}

	__syncthreads();

	shuffled[my_cell_id] = tmp[my_cell_id];
}

/**
 * Prints the passed array like a sudoku puzzle in ascii art
 * @numbers array to print
 */
 void sudoku_print(unsigned int* numbers)
{
  int i;
  int j;
  int block_dim = round(sqrt(DIM));

  printf("\n_________________________________________\n");

  for (i = 0; i < DIM; i++) {

    printf("||");
    for (j = 0; j < DIM; j++) {
      printf(" %u |", numbers[ ( (i*DIM) + j ) ]);
      if((j+1) % block_dim == 0) {
          printf("|");
      }
    }

    j = 0;
    //Breaks between each row
    if( ((i+1) % block_dim) == 0) {
      printf("\n||___|___|___||___|___|___||___|___|___||\n");
    } else {
      //TODO:make this able to handle other sizes prettily
      printf("\n||---|---|---||---|---|---||---|---|---||\n");
    }
 }
}

void main_sub()
{
  /* Calculate the size of the array */
  int array_size_in_bytes = (sizeof(unsigned int) * (CELLS));

  unsigned int h_puzzle[CELLS] = {0,0,4,3,0,0,2,0,9,
																0,0,5,0,0,9,0,0,1,
																0,7,0,0,6,0,0,4,3,
																0,0,6,0,0,2,0,8,7,
																1,9,0,0,0,7,4,0,0,
																0,5,0,0,8,3,0,0,0,
																6,0,0,0,0,0,1,0,5,
																0,0,3,5,0,8,6,9,0,
																0,4,2,9,1,0,3,0,0};
	unsigned int *h_pinned_puzzle;
  unsigned int *h_solution;

  //pin it
  hipHostMalloc((void **)&h_pinned_puzzle, array_size_in_bytes, hipHostMallocDefault);
  hipHostMalloc((void **)&h_solution, array_size_in_bytes, hipHostMallocDefault);

	// Copy it to pinned memory
	memcpy(h_pinned_puzzle, h_puzzle, array_size_in_bytes);

  /* Declare and allocate pointers for GPU based parameters */
  unsigned int *d_puzzle;
  unsigned int *d_solution;

  hipMalloc((void **)&d_puzzle, array_size_in_bytes);
  hipMalloc((void **)&d_solution, array_size_in_bytes);

  /* Copy the CPU memory to the GPU memory */
  hipMemcpy(d_puzzle, h_pinned_puzzle, array_size_in_bytes, hipMemcpyHostToDevice);

  /* Designate the number of blocks and threads */
  const unsigned int num_blocks = CELLS/THREADS_PER_BLOCK;
  const unsigned int num_threads = CELLS/num_blocks;

  /* Execute the kernel and keep track of start and end time for duration */
  float duration = 0;

  hipEvent_t start_time = get_time();

	solve<<<num_blocks, num_threads>>>(d_puzzle, d_solution);

  hipEvent_t end_time = get_time();
  hipEventSynchronize(end_time);

  hipEventElapsedTime(&duration, start_time, end_time);

  /* Copy the changed GPU memory back to the CPU */
  hipMemcpy(h_solution, d_solution, array_size_in_bytes, hipMemcpyDeviceToHost);

	//TODO: would like puzzle and solution to be able to print side by side
	printf("Puzzle:\n");
	sudoku_print(h_puzzle);

	printf("Solution:\n");
  sudoku_print(h_solution);

	printf("\tSolved in: %fmsn\n", duration);

  /* Free the GPU memory */
  hipFree(d_puzzle);
  hipFree(d_solution);

  /* Free the pinned CPU memory */
  hipHostFree(h_pinned_puzzle);
	hipHostFree(h_solution);
}

/**
 * Prints the correct usage of this file
 * @name is the name of the executable (argv[0])
 */
void print_usage(char *name)
{
  printf("Usage: %s \n", name);
}

/**
 * Entry point for execution. Checks command line arguments
 * then passes execution to subordinate function
 */
int main(int argc, char *argv[])
{
  /* Check the number of arguments, print usage if wrong */
  if(argc != 1) {
    printf("Error: Incorrect number of command line arguments\n");
    print_usage(argv[0]);
    exit(-1);
  }

  printf("\n");

  main_sub();

  return EXIT_SUCCESS;
}