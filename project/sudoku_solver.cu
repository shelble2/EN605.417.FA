#include "hip/hip_runtime.h"
/**
 * sudoku_solver.cu
 * Sarah Helble
 * 2017-12-07
 *
 * Top-level file for the sudoku solver project.
 * This includes the main function but calls out to the sudoku_utils and
 * solver_kernels for most operations.
 */

#include <stdio.h>
#include <stdlib.h>

#include "sudoku_utils.cuh"
#include "solver_kernels.cuh"

int execute_kernel_two_loop(unsigned int *hp_puzzles, int cells, unsigned int **solutions)
{
	int count = 0;
	int array_size_in_bytes = (sizeof(unsigned int)*(cells*2));
	hipError_t cuda_ret;
	*solutions = NULL;

	unsigned int *d_puzzles;
	unsigned int *d_solutions;
	//XXX: could have just passed the right total number of cells, and used that
	// to find the number of puzzles to pass as number of blocks
	cuda_ret = hipMalloc((void **)&d_puzzles, array_size_in_bytes);
	if(cuda_ret != hipSuccess) {
		printf("ERROR in hipMalloc for d_puzzle\n");
		count = -1;
		goto malloc_puzzle_error;
	}
	cuda_ret = hipMalloc((void **)&d_solutions, array_size_in_bytes);
	if(cuda_ret != hipSuccess) {
		printf("ERROR in hipMalloc for d_solution\n");
		count = -1;
		goto malloc_solution_error;
	}

	// While the puzzle is not finished, iterate until LOOP_LIMIT is reached
	do {

		/* Copy the CPU memory to the GPU memory */
		cuda_ret = hipMemcpy(d_puzzles, hp_puzzles, array_size_in_bytes,
									hipMemcpyHostToDevice);
		if(cuda_ret != hipSuccess) {
			printf("ERROR memcpy host to device (%d)\n", cuda_ret);
			count = -1;
			goto memcpy_error;
		}

		printf("Iteration %d\n", count);
		solve_mult_by_possibility<<<2, cells>>>(d_puzzles, d_solutions);

		/* Copy the changed GPU memory back to the CPU */
		cuda_ret = hipMemcpy(hp_puzzles, d_solutions, array_size_in_bytes,
						hipMemcpyDeviceToHost);
		if(cuda_ret != hipSuccess) {
			printf("ERROR memcpy device to host (%d)\n", cuda_ret);
			count = -1;
			goto memcpy_error;
		}

		count = count + 1;
	} while ((check_if_done(hp_puzzles) == 1) && (count <= LOOP_LIMIT));

	if(count == LOOP_LIMIT) {
		printf("[ WARNING ] Could not find a solution within max allowable (%d) iterations.\n", LOOP_LIMIT);
	}

	*solutions = hp_puzzles;

memcpy_error:
	hipFree(d_solutions);
malloc_solution_error:
	hipFree(d_puzzles);
malloc_puzzle_error:
	return count;
}

/**
 * Loops over the kernel until the puzzle is solved or LOOP_LIMIT is reached
 * On success, returns the number of iterations performed, and **solution is
 * set to the end result. Returns -1 on failure.
 * hp_puzzle is the host-pinned puzzle of unsigned ints
 * cells is the number of cells in the puzzle
 */
int execute_kernel_loop(unsigned int *hp_puzzle, int cells, unsigned int **solution)
{
	int count = 0;
	int array_size_in_bytes = (sizeof(unsigned int) * (cells));
	hipError_t cuda_ret;
	*solution = NULL;

	/* Declare and allocate pointers for GPU based parameters */
	unsigned int *d_puzzle;
	unsigned int *d_solution;
	cuda_ret = hipMalloc((void **)&d_puzzle, array_size_in_bytes);
	if(cuda_ret != hipSuccess) {
		printf("ERROR in hipMalloc for d_puzzle\n");
		count = -1;
		goto malloc_puzzle_error;
	}
	cuda_ret = hipMalloc((void **)&d_solution, array_size_in_bytes);
	if(cuda_ret != hipSuccess) {
		printf("ERROR in hipMalloc for d_solution\n");
		count = -1;
		goto malloc_solution_error;
	}

	hipStream_t stream;
	hipStreamCreate(&stream);

	// While the puzzle is not finished, iterate until LOOP_LIMIT is reached
	do {
		/* Copy the CPU memory to the GPU memory */
		cuda_ret = hipMemcpyAsync(d_puzzle, hp_puzzle, array_size_in_bytes,
									hipMemcpyHostToDevice, stream);
		if(cuda_ret != hipSuccess) {
			printf("ERROR memcpy host to device\n");
			count = -1;
			goto memcpy_error;
		}

		solve_by_possibility<<<1, cells>>>(d_puzzle, d_solution);

		/* Copy the changed GPU memory back to the CPU */
		cuda_ret = hipMemcpyAsync(hp_puzzle, d_solution, array_size_in_bytes,
						hipMemcpyDeviceToHost, stream);
		if(cuda_ret != hipSuccess) {
			printf("ERROR memcpy host to device\n");
			count = -1;
			goto memcpy_error;
		}

		hipStreamSynchronize(stream);

		count = count + 1;
	} while ((check_if_done(hp_puzzle) == 1) && (count <= LOOP_LIMIT));

	if(count == LOOP_LIMIT) {
		printf("[ WARNING ] Could not find a solution within max allowable (%d) iterations.\n", LOOP_LIMIT);
	}

	*solution = hp_puzzle;

memcpy_error:
	hipFree(d_solution);
malloc_solution_error:
	hipFree(d_puzzle);
malloc_puzzle_error:
	return count;
}

/**
 * Solves two puzzles at once (one per block)
 */
 int solve_two_puzzles(unsigned int *h_puzzles, int cells, FILE *metrics_fd, int verbosity)
 {
	 printf("in solve two puzzles\n");
	 int ret = 0;
	 int array_size_in_bytes = (sizeof(unsigned int) * (cells *2));
	 hipError_t cuda_ret;

	 //pin it and copy to pinned memory
	 unsigned int *h_pinned_puzzles;
	 unsigned int *solutions;
	 cuda_ret = hipHostMalloc((void **) &h_pinned_puzzles, array_size_in_bytes);
	 if(cuda_ret != hipSuccess) {
 		printf("Error mallocing pinned host memory\n");
 		return -1;
 	}
 	memcpy(h_pinned_puzzles, h_puzzles, array_size_in_bytes);

 	if(verbosity == 1) {
 		sudoku_print_two(h_puzzles);
 	}

 	/* Execute the kernel and keep track of start and end time for duration */
 	float duration = 0;
 	hipEvent_t start_time = get_time();

 	int count = execute_kernel_two_loop(h_pinned_puzzles, cells, &solutions);
 	if(count <= 0) {
 		printf("ERROR: returned %d from execute_kernel_loop\n", count);
 		hipHostFree(h_pinned_puzzles);
 		return -1;
 	}

 	hipEvent_t end_time = get_time();
 	hipEventSynchronize(end_time);
 	hipEventElapsedTime(&duration, start_time, end_time);

 	if(verbosity == 1) {
 		sudoku_print_two(h_pinned_puzzles);
 		printf("\tSolved in %d increments and %fms\n", count, duration);
 	}

 	//XXX: Could this print to file be a bottleneck?
 	if(metrics_fd != NULL) {
 		output_metrics_to_file(metrics_fd, h_puzzles, h_pinned_puzzles, count, duration);
 	}

 	/* Free the pinned CPU memory */
 	hipHostFree(h_pinned_puzzles);
 	return ret;
 }

/**
 * Solves the passed puzzle
 * h_puzzle is the host array of ints that form the puzzle,
 * cells is the number of cells in the puzzle
 * metrics_fd is an open file descriptor to the output file
 * verbosity is a flag for extra prints. If == 1, will print every puzzle and
 * solution to STDOUT. Otherwise, will just print batch metrics. Either way,
 * metrics for each specific puzzle can be found in output file
 */
int solve_puzzle(unsigned int *h_puzzle, int cells, FILE *metrics_fd, int verbosity)
{
	int ret = 0;
	int array_size_in_bytes = (sizeof(unsigned int) * (cells));
	hipError_t cuda_ret;

	//pin it and copy to pinned memory
	unsigned int *h_pinned_puzzle;
	unsigned int *solution;
	cuda_ret = hipHostMalloc((void **)&h_pinned_puzzle, array_size_in_bytes);
	if(cuda_ret != hipSuccess) {
		printf("Error mallocing pinned host memory\n");
		return -1;
	}
	memcpy(h_pinned_puzzle, h_puzzle, array_size_in_bytes);

	if(verbosity == 1) {
		printf("Puzzle:\n");
		sudoku_print(h_puzzle,0);
	}

	/* Execute the kernel and keep track of start and end time for duration */
	float duration = 0;
	hipEvent_t start_time = get_time();

	int count = execute_kernel_loop(h_pinned_puzzle, cells, &solution);
	if(count <= 0) {
		printf("ERROR: returned %d from execute_kernel_loop\n", count);
		hipHostFree(h_pinned_puzzle);
		return -1;
	}

	hipEvent_t end_time = get_time();
	hipEventSynchronize(end_time);
	hipEventElapsedTime(&duration, start_time, end_time);

	if(verbosity == 1) {
		printf("Solution:\n");
		sudoku_print(h_pinned_puzzle, 0);
		printf("\tSolved in %d increments and %fms\n", count, duration);
	}

	//XXX: Could this print to file be a bottleneck?
	if(metrics_fd != NULL) {
		output_metrics_to_file(metrics_fd, h_puzzle, h_pinned_puzzle, count, duration);
	}

	/* Free the pinned CPU memory */
	hipHostFree(h_pinned_puzzle);
	return ret;
}

/**
 * Find the best available device for our use case and set it
 * Right now, this just picks the one with the highest number of
 * multiprocessors.
 */
void find_and_select_device()
{
	printf("----------------------------------------------\n");
	printf("Finding the best device for the job\n");
	int num_devices;
	int device = 0;
	int max_mp = 0;
	int i;

	// Figure out how many devices there are
	hipGetDeviceCount(&num_devices);
	printf("%d possible devices\n", num_devices);

	printf("Selecting the one with the highest number of multiprocessors\n");
	for(i = 0; i < num_devices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		if(prop.multiProcessorCount > max_mp) {
			max_mp = prop.multiProcessorCount;
			device = i;
		}
		printf("Device %d has : \n\t%d multiprocessors\n\t%d warp size\n",
				i, prop.multiProcessorCount, prop.warpSize);
	}

	printf("Selected device %d\n", device);
	hipSetDevice(device);
	printf("----------------------------------------------\n");
}

void solve_from_fp_two(FILE *input_fp, FILE *metrics_fp, int verbosity,
						int *solved, int *unsolvable, int *errors)
{
	char *line1 = NULL;
	char *line2 = NULL;
	size_t len = 0;

	int tmp_solved = 0;
	int tmp_errors = 0;
	int tmp_unsolvable = 0;
	int ret;

	while(getline(&line1, &len, input_fp) != -1) {
		unsigned int *h_puzzle;

		ret = getline(&line2, &len, input_fp);

		// Cop out if another line's not there
		if(ret == -1) {
			h_puzzle = load_puzzle(line1, CELLS);
			ret = solve_puzzle(h_puzzle, CELLS, metrics_fp, verbosity);
			goto take_count;
		}

		h_puzzle = load_two_puzzles(line1, line2, CELLS);
		ret = solve_two_puzzles(h_puzzle, CELLS, metrics_fp, verbosity);

take_count:
		// Keep track of the statuses coming out
		if(ret == -1) {
			tmp_errors = tmp_errors + 1;
		} else if(ret == LOOP_LIMIT) {
			tmp_unsolvable = tmp_unsolvable + 1;
		} else {
			tmp_solved = tmp_solved + 1;
		}
	}

	*solved = tmp_solved;
	*unsolvable = tmp_unsolvable;
	*errors = tmp_errors;
}

/**
 * Loads the lines from the open file descriptor one by one and solves them
 * input_fp is the open file descriptor to read from
 * metrics_fp is an open file descriptor to write metrics to
 * Does not return a value, but sets solved to the number of puzzles
 * successfully finished, unsolved to the number that could not be Solved within
 * the LOOP_LIMIT, and sets error to the number of puzzles that returned with
 * error
 */
 // TODO: needs a better name
void solve_from_fp_one(FILE *input_fp, FILE *metrics_fp, int verbosity,
						int *solved, int *unsolvable, int *errors)
{
	char *line = NULL;
	size_t len = 0;

	int tmp_solved = 0;
	int tmp_errors = 0;
	int tmp_unsolvable = 0;
	int ret;

	while(getline(&line, &len, input_fp) != -1) {
		unsigned int *h_puzzle = load_puzzle(line, CELLS);
		ret = solve_puzzle(h_puzzle, CELLS, metrics_fp, verbosity);

		// Keep track of the statuses coming out
		if(ret == -1) {
			tmp_errors = tmp_errors + 1;
		} else if(ret == LOOP_LIMIT) {
			tmp_unsolvable = tmp_unsolvable + 1;
		} else {
			tmp_solved = tmp_solved + 1;
		}
	}

	*solved = tmp_solved;
	*unsolvable = tmp_unsolvable;
	*errors = tmp_errors;
}

/**
 * Entry point for execution. Checks command line arguments
 * then passes execution to subordinate function
 */
int main(int argc, char *argv[])
{
	int verbosity = 1;
	if(argc != 2 && argc != 3) {
		printf("Error: Incorrect number of command line arguments\n");
		printf("Usage: %s [input_file] (v=0)\n", argv[0]);
		exit(-1);
	}
	printf("\n");

	char *input_fn = argv[1];
	FILE *input_fp = fopen(input_fn, "r");
	if(input_fp == NULL) {
		printf("Failed to open input file %s\n", input_fn);
		return -1;
	}

	// TODO: this would be prettier if switched to optparse
	if((argc == 3) && (strcmp(argv[2], "v=0") == 0)) {
		verbosity = 0;
	}

	//TODO: make this a command line option instead of Hardcoded
	char *metrics_fn = "metrics.csv";
	FILE *metrics_fp = fopen(metrics_fn, "w");
	if(metrics_fp == NULL) {
		printf("Failed to open metrics file for writing\n");
		fclose(input_fp);
		return -1;
	}

	find_and_select_device();

	/* Keep track of total duration */
	float duration = 0;
	hipEvent_t start_time = get_time();

	int solved;
	int unsolvable;
	int errors;
/*	solve_from_fp_one(input_fp, metrics_fp, verbosity,
						&solved, &unsolvable, &errors);
*/
	solve_from_fp_two(input_fp, metrics_fp, verbosity,
						&solved, &unsolvable, &errors);

	hipEvent_t end_time = get_time();
	hipEventSynchronize(end_time);
	hipEventElapsedTime(&duration, start_time, end_time);

	printf("\nFrom a dataset of %d puzzles,\n", solved + unsolvable + errors);
	printf("Solved %d, partially solved %d, and encountered %d errors in %0.3fms\n\n", solved, unsolvable, errors, duration);
	printf("Individual puzzle data output to %s\n", metrics_fn);

	fclose(input_fp);
	fclose(metrics_fp);

	return EXIT_SUCCESS;
}
